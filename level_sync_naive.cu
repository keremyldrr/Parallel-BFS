
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <limits.h>
#include <vector>
#include <queue>          // std::queue
#include <chrono>

using namespace std;
int *row;
int *col;
int *path;
bool *visited;
int N;


__global__ void processLevel(int *row, int *col, int *path, int level, int N, int *check, unsigned int tnum, int chunk) {
 unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id == 0)
    *check = 0; // reset value
  // __syncthreads();  
  for(unsigned int i = id; i < N; i += tnum) {
    if(path[i] == level) {
      int start = row[i];
      int stop = row[i+1];
      int w;
      for(int j = start; j < stop; j++) {
	w = col[j];
	if(path[w] > level)
	  path[w] = level + 1;
      }
    }
  }
  __syncthreads();
  for(unsigned int i = id; i < N; i += tnum) {
    if(path[i] == INT_MAX) {
      atomicOr(check,1);
      break;
    }
  }
}

int main(int argc, const char **argv) {

  if(argc != 2) {
    cout << "wrong input " << endl;
    return 0;
  }


  srand (time(NULL));

  // tested on coPapersDBLP.mtx
  string filename = argv[1]; //"coPapersDBLP.mtx"; // edit
  //string filepath = "/home/cema/Documents/CS406/project/data/" + filename;
  ifstream file(filename.c_str());

  if(file.fail()) {
    cout << "Path not found" << endl;
    return 0;
  }

  string line;
  getline(file,line);

  stringstream s(line);
  bool symmetric = false;
  string word;
  while(s >> word) // check for symmetric
    if(word == "symmetric")
      symmetric = true;

  while(file.peek() == '%')//dirty boi
    file.ignore(2048, '\n');

  int  M, edge;
  file >> N >> M >> edge;



  cout << "Graph has " << N << " nodes and " << (1+symmetric*1)*edge << " edges and symmetric " << symmetric << endl;

  int n;
  bool based0 = false;
  while(file >> n) {
    if (n == 0) {
      based0 = true;
      break;
    }
  }
  file.close();




  if(based0)
    cout << "Graph is 0 based" << endl;
  else
    cout << "Graph is 1 based and is being turned in to 0 base." << endl;

  file.open(filename.c_str());
  while(file.peek() == '%')
    file.ignore(2048, '\n');
  file >> n >> n >> n; //tmp


  int i,j;

  vector<vector<int> > v(N, vector<int>(0));

  for(int k = 0; k < edge; k++) {
    file >> i >> j;
    if(!based0) { // make 0 based;
      i--;
      j--;
    }

    v[i].push_back(j);
    if(symmetric)
      v[j].push_back(i);

  }
  file.close();

  edge *= 1 + (int)symmetric;


  row = new int[N+1];
  col = new int[edge];

  row[0] = 0;
  int index = 0;
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < v[i].size(); j++) {
      col[index] = v[i][j];
      index++;
    }
    row[i+1] = index;
  }

  path = new int[N];
  visited = new bool[N];
  //    int *levels = new int[N];
  for(int i = 0; i < N; i++)
    path[i] =  INT_MAX;
  cout << "Preprocessing complete " << endl;


  // algorithm starts here
  //*************************************************
  hipSetDevice(0);
  int root = rand() % (N-1);
  root = 0;
  path[root] = 0;
  // copy graph to device
  int *d_row;
  int *d_col;
  int *d_path;
  int *d_check;
  int size_row = (N+1)*sizeof(int);
  int size_col = edge*sizeof(int);
  int size_path = N*sizeof(int);
   
  hipMalloc((void **)&d_row,size_row);
  hipMalloc((void **)&d_col,size_col);
  hipMalloc((void **)&d_path,size_path);
  hipMalloc((void **)&d_check,sizeof(int));
  
  cout << "Memory allocated" <<endl;
  
  hipMemcpy(d_row,row,size_row,hipMemcpyHostToDevice);
  hipMemcpy(d_col,col,size_col,hipMemcpyHostToDevice);
  hipMemcpy(d_path,path,size_path,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  
  cout << "Graph copied to device" << endl;
  unsigned int numthread = 1024;
  unsigned int numblock = 2*4096;
  unsigned int tnum = numblock*numthread;
  int chunk = N/tnum;
  if(chunk*tnum < N)
    chunk += 1;
  if(tnum >= N) {
    chunk = 1;
    numblock = ceil((double)N/numthread);
  }
  int check = 1;
  int level = 0;
  cout << "starting" << endl;
  auto start = chrono::high_resolution_clock::now();
  while(check == 1) {
    processLevel<<<numblock,numthread>>>(d_row,d_col,d_path,level,N,d_check,tnum,chunk);
    hipDeviceSynchronize();
    hipMemcpy(&check,d_check,sizeof(int),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    level++;
  }
  auto finish = chrono::high_resolution_clock::now();
  

  hipMemcpy(path,d_path,size_path,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int count = 0;
  for(int i = 0; i < N; i++) { 
    if(path[i] == INT_MAX) {
      count++;
      //cout << i << endl;
    }
  }
  
  if(count == 0)
    cout << "BFS Complete" << endl;
  else
    cout << count << " unvisited nodes" << endl;

  chrono::duration<double> elapsed = finish - start;
  cout << "Time: " << elapsed.count() << endl; 

  hipFree(d_row);
  hipFree(d_col);
  hipFree(d_path);

  delete[] col;
  delete[] row;
  //delete[] path;

  cout << "Graph deleted" << endl;

  ofstream ofile;
  ofile.open("results.txt");
  for(int i = 0; i < N; i++)
    ofile << path[i] << "\n";
  ofile.close();

  delete[] path;
  return 0;
}
