
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <limits.h>
#include <vector>
#include <queue>          // std::queue


using namespace std;
int *row;
int *col;
int *path;
bool *visited;
int N;


__global__ void top_down_step(int *load,int *path,int dist,int *row,int *col,int *frontier,int *next,int size)
{
unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x; // index in frontier
int sum = 0;

 if(ind < size)
   {
     if ((threadIdx.x | threadIdx.y | threadIdx.z
	  | blockIdx.x | blockIdx.y | blockIdx.z) == 0) {
       // statements that are executed once per kernel invocation,
       // at an unspecified time
       *load = 0;     
     }
     
     __syncthreads();
     int elem = frontier[ind];
     int start = row[elem];
     int end = row[elem+1];
     
     for(int I = start; I < end; I++)
       {
	 int i = col[I];
	 //OPTIMIZATION
	 if(path[i] == -1)
	   {
	     //*load = 1222;  
	     path[i] = dist+1;
	     int t = atomicAdd(load,1);
	     next[t] = i;
	   }
	  
	}
  }
 
    
}


int main(int argc, const char **argv) {

  if(argc != 2) {
    cout << "wrong input " << endl;
    return 0;
  }


  srand (time(NULL));

  // tested on coPapersDBLP.mtx
  string filename = argv[1]; //"coPapersDBLP.mtx"; // edit
  //string filepath = "/home/cema/Documents/CS406/project/data/" + filename;
  ifstream file(filename.c_str());

  if(file.fail()) {
    cout << "Path not found" << endl;
    return 0;
  }

  string line;
  getline(file,line);

  stringstream s(line);
  bool symmetric = false;
  string word;
  while(s >> word) // check for symmetric
    if(word == "symmetric")
      symmetric = true;

  while(file.peek() == '%')//dirty boi
    file.ignore(2048, '\n');

  int  M, edge;
  file >> N >> M >> edge;



  cout << "Graph has " << N << " nodes and " << (1+symmetric*1)*edge << " edges and symmetric " << symmetric << endl;

  int n;
  bool based0 = false;
  while(file >> n) {
    if (n == 0) {
      based0 = true;
      break;
    }
  }
  file.close();




  if(based0)
    cout << "Graph is 0 based" << endl;
  else
    cout << "Graph is 1 based and is being turned in to 0 base." << endl;

  file.open(filename.c_str());
  while(file.peek() == '%')
    file.ignore(2048, '\n');
  file >> n >> n >> n; //tmp


  int i,j;

  vector<vector<int> > v(N, vector<int>(0));

  for(int k = 0; k < edge; k++) {
    file >> i >> j;
    if(!based0) { // make 0 based;
      i--;
      j--;
    }

    v[i].push_back(j);
    if(symmetric)
      v[j].push_back(i);

  }
  file.close();

  edge *= 1 + (int)symmetric;


  row = new int[N+1];
  col = new int[edge];

  row[0] = 0;
  int index = 0;
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < v[i].size(); j++) {
      col[index] = v[i][j];
      index++;
    }
    row[i+1] = index;
  }

  path = new int[N];
  
  int *frontier = new int[N];
  int *next = new int[N];
  //    int *levels = new int[N];
  for(int i = 0; i < N; i++)
    {
      path[i] =  -1;//INT_MAX;
      frontier[i] = -1;
      next[i] = -1;
    }

 
 cout << "Preprocessing complete " << endl;


  // algorithm starts here
  //*************************************************
  hipSetDevice(0);
  int root = rand() % (N-1);
  root = 0;
  path[root] = 0;
  // copy graph to device
  int *d_row;
  int *d_col;
  int *d_path;
  int *d_load;
  int *d_front;
  int *d_next;
  int size_row = (N+1)*sizeof(int);
  int size_col = edge*sizeof(int);
  int size_path = N*sizeof(int);

  
  int frontSize = size_path;

  // for the row check up
  hipMalloc((void **)&d_row,size_row);
  hipMalloc((void **)&d_col,size_col);
  // path array
  hipMalloc((void **)&d_path,size_path);
  //frontier on gpu
  hipMalloc((void **)&d_front,frontSize);
  hipMalloc((void **)&d_next,frontSize);
  // integer for check operation
  hipMalloc((void **)&d_load,sizeof(int)); // load is the frontier size 

  int load = 1;
  frontier[0] = root;
  cout << "Memory allocated" <<endl;
  // move memory
  hipMemcpy(d_row,row,size_row,hipMemcpyHostToDevice);
  hipMemcpy(d_col,col,size_col,hipMemcpyHostToDevice);
  hipMemcpy(d_path,path,size_path,hipMemcpyHostToDevice);
  hipMemcpy(d_front,frontier,frontSize,hipMemcpyHostToDevice);
  hipMemcpy(d_next,next,frontSize,hipMemcpyHostToDevice);
  hipMemcpy(d_load,&load,sizeof(int),hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  cout << "Graph copied to device" << endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned int threads = prop.maxThreadsPerBlock;
  unsigned int threadDIM = prop.maxThreadsDim[3];

  cout << "Here are the specs\n";
  cout << "The threads: " << threads << "\n";
  cout << "Thread DIM:  " << threadDIM << "\n";
  int BlockDIM = int(ceil(double(N)/threads));  // BlockDIM ->> number of blocks necessary

  int level = 0;
  cout << "Algo starts now... Hold on to your seats" << endl;
  
  double start = omp_get_wtime();
  int zeroman = 0;
  //BlockDIM = 1;
  cout << "init load " << load << endl;
  int size = load;

  //threads = load;
  int m = 0;
  
  while(load != 0)
    {
      //double a = omp_get_wtime();
      //    cudaMemcpy(d_load,&zeroman,sizeof(int),cudaMemcpyHostToDevice);
      //      fs+=omp_get_wtime()-a;
      //double b = omp_get_wtime();
      top_down_step<<<BlockDIM,1024>>>(d_load,d_path,level,d_row,d_col,d_front,d_next,size);
      //sc+=omp_get_wtime()-b;
      //cudaDeviceSynchronize();
      int *tmp = d_front;
      d_front = d_next;
      d_next = tmp;

      
      hipMemcpy(&load,d_load,sizeof(int),hipMemcpyDeviceToHost);
      //      cudaDeviceSynchronize();
      size = load;
      BlockDIM = int(ceil(double(size)/1024));
      level++;
      
    }
  
  double finish = omp_get_wtime();
  hipMemcpy(path,d_path,size_path,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int count = 0;
  for(int i = 0; i < N; i++) {
    if(path[i] == -1) {
      count++;
    }
  }
  //cout << "MEMCPY1 " << fs << " "; 
  if(count == 0)
    cout << "BFS Complete" << endl;
  else
    cout << count << " unvisited nodes" << endl;

  double elapsed = finish - start;
  cout << "Time: " << elapsed << endl;

  hipFree(d_row);
  hipFree(d_col);
  hipFree(d_path);

  delete[] col;
  delete[] row;
  //delete[] path;

  cout << "Graph deleted" << endl;

  ofstream ofile;
  ofile.open("results.txt");
  for(int i = 0; i < N; i++)
    ofile << path[i] << "\n";
  ofile.close();

  delete[] path;
  return 0;
}
