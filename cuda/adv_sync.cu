
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <limits.h>
#include <vector>
#include <queue>          // std::queue


using namespace std;
int *row;
int *col;
int *path;
bool *visited;
int N;


__global__ void top_down_step(int *load,int *path,int dist,int *row,int *col)
{
  unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x;

    if(path[ind] == dist)
      {
      int start = row[ind];
      int end = row[ind+1];
      for(int I = start; I < end; I++)
	{
	  int i = col[I];
	  if(path[i] == -1)
	    {
	      path[i] = dist+1;
	      //*load = 1;
	      atomicAdd(load, 1);
	    }
	}
      }
}

__global__ void bottom_up_step(int *load,int *path,int dist,int *row,int *col)
{
  unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x;
  if(path[ind] == -1)
      {
      int start = row[ind];
      int end = row[ind+1];
      for(int I = start; I < end; I++)
	{
	  int i = col[I];
	  if(path[i] == dist)
	    {
	      path[ind] = dist+1;
	      atomicAdd(load, 1);
	      break;
	    }
	  
	}
      }
}


int main(int argc, const char **argv) {

  if(argc != 2) {
    cout << "wrong input " << endl;
    return 0;
  }


  srand (time(NULL));

  // tested on coPapersDBLP.mtx
  string filename = argv[1]; //"coPapersDBLP.mtx"; // edit
  //string filepath = "/home/cema/Documents/CS406/project/data/" + filename;
  ifstream file(filename.c_str());

  if(file.fail()) {
    cout << "Path not found" << endl;
    return 0;
  }

  string line;
  getline(file,line);

  stringstream s(line);
  bool symmetric = false;
  string word;
  while(s >> word) // check for symmetric
    if(word == "symmetric")
      symmetric = true;

  while(file.peek() == '%')//dirty boi
    file.ignore(2048, '\n');

  int  M, edge;
  file >> N >> M >> edge;



  cout << "Graph has " << N << " nodes and " << (1+symmetric*1)*edge << " edges and symmetric " << symmetric << endl;

  int n;
  bool based0 = false;
  while(file >> n) {
    if (n == 0) {
      based0 = true;
      break;
    }
  }
  file.close();




  if(based0)
    cout << "Graph is 0 based" << endl;
  else
    cout << "Graph is 1 based and is being turned in to 0 base." << endl;

  file.open(filename.c_str());
  while(file.peek() == '%')
    file.ignore(2048, '\n');
  file >> n >> n >> n; //tmp


  int i,j;
  vector<vector<int> > v_inv(N, vector<int>(0));
  vector<vector<int> > v(N, vector<int>(0));

  for(int k = 0; k < edge; k++) {
    file >> i >> j;
    if(!based0) { // make 0 based;
      i--;
      j--;
    }

    v[i].push_back(j);
    v_inv[j].push_back(i); // inverse graph
    if(symmetric)
      v[j].push_back(i);

  }
  file.close();

  edge *= 1 + (int)symmetric;

  int *row_inv = new int[N+1];
  int *col_inv = new int[edge];
  
  row = new int[N+1];
  col = new int[edge];

  row[0] = 0;
  int index = 0;
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < v[i].size(); j++) {
      col[index] = v[i][j];
      index++;
    }
    row[i+1] = index;
  }

  //inv graph porcessing
  row_inv[0] = 0;
  index = 0;
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < v_inv[i].size(); j++) {
      col_inv[index] = v_inv[i][j];
      index++;
    }
    row_inv[i+1] = index;
  }

  path = new int[N];
  visited = new bool[N];
  //    int *levels = new int[N];
  for(int i = 0; i < N; i++)
    path[i] =  -1;//INT_MAX;
 

 
 cout << "Preprocessing complete " << endl;


  // algorithm starts here
  //*************************************************
  hipSetDevice(0);
  int root = rand() % (N-1);
  root = 0;
  path[root] = 0;

  int root_begin = row[root];
  int root_end = row[root+1];
  int root_begin_inv = row_inv[root];
  int root_end_inv = row_inv[root+1];
  
  for(int I = root_begin; I < root_end; I++){
	path[col[I]] = 1;
  }
  // copy graph to device
  int *d_row;
  int *d_col;
  int *d_row_inv;
  int *d_col_inv;
  int *d_path;
  int *d_load;
  int size_row = (N+1)*sizeof(int);
  int size_col = edge*sizeof(int);
  int size_path = N*sizeof(int);
  int load = 0;
  

  // for the row check up
  hipMalloc((void **)&d_row,size_row);
  hipMalloc((void **)&d_col,size_col);
  hipMalloc((void **)&d_row_inv,size_row);
  hipMalloc((void **)&d_col_inv,size_col);
  // path array
  hipMalloc((void **)&d_path,size_path);
  // integer for check operation
  hipMalloc((void **)&d_load,sizeof(int)); 
  
  cout << "Memory allocated" <<endl;
  // move memory
  hipMemcpy(d_row,row,size_row,hipMemcpyHostToDevice);
  hipMemcpy(d_col,col,size_col,hipMemcpyHostToDevice);
  hipMemcpy(d_path,path,size_path,hipMemcpyHostToDevice);
  hipMemcpy(d_load,&load,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(d_row_inv,row_inv,size_row,hipMemcpyHostToDevice);
  hipMemcpy(d_col_inv,col_inv,size_col,hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  cout << "Graph copied to device" << endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned int threads = prop.maxThreadsPerBlock;
  unsigned int threadDIM = prop.maxThreadsDim[3];

  cout << "Here are the specs\n";
  cout << "The threads: " << threads << "\n";
  cout << "Thread DIM:  " << threadDIM << "\n";
  int BlockDIM = int(ceil(double(N)/threads));  // BlockDIM ->> number of blocks necessary

  int level = 1;
  cout << "Algo starts now... Hold on to your seats" << endl;
  load = 1;
  double start = omp_get_wtime();
  int zeroman = 0;
  while(load != 0)
    {

      hipMemcpy(d_load,&zeroman,sizeof(int),hipMemcpyHostToDevice);
      if(double(load)/N <0.1)
	{
	  //cout << "topdown " << load << endl;
      top_down_step<<<BlockDIM,threads>>>(d_load,d_path,level,d_row,d_col);
	}
      else
	{
	  //cout << "bottomup " << load << endl;
	  if(symmetric)
	    bottom_up_step<<<BlockDIM,threads>>>(d_load,d_path,level,d_row,d_col);
	  else
	    bottom_up_step<<<BlockDIM,threads>>>(d_load,d_path,level,d_row_inv,d_col_inv);
	}
      hipMemcpy(&load,d_load,sizeof(int),hipMemcpyDeviceToHost);
      level++;
    }
  
  double finish = omp_get_wtime();
  

  hipMemcpy(path,d_path,size_path,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int count = 0;
  for(int i = 0; i < N; i++) {
    if(path[i] == -1) {
      count++;
    }
  }

  if(count == 0)
    cout << "BFS Complete" << endl;
  else
    cout << count << " unvisited nodes" << endl;

  double elapsed = finish - start;
  cout << "Time: " << elapsed << endl;

  hipFree(d_row);
  hipFree(d_col);
  hipFree(d_path);
  hipFree(d_row_inv);
  hipFree(d_col_inv);

  delete[] col;
  delete[] row;
  delete[] col_inv;
  delete[] row_inv;
  //delete[] path;

  cout << "Graph deleted" << endl;

  ofstream ofile;
  ofile.open("results.txt");
  for(int i = 0; i < N; i++)
    ofile << path[i] << "\n";
  ofile.close();

  delete[] path;
  return 0;
}
