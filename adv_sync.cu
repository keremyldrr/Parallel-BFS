#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <limits.h>
#include <vector>
#include <queue>          // std::queue
#include <chrono>

using namespace std;
int *row;
int *col;
int *path;
bool *visited;
int N;


__global__ void top_down_step(int *load,int *path,int dist,int *row,int *col) {
  unsigned int ind = threadIdx.x + blockIdx.x * blockDim.x;
  *load = 0;
  if(path[i] == dist){
  int start = row[ind];
  int end = row[ind+1];
  for(int I = start; I < end; I++){
      if(path[col[I]] == dist)
        {
          path[col[I]] = dist+1;
          *load = 1;
        }
  }
 }
}


int main(int argc, const char **argv) {

  if(argc != 2) {
    cout << "wrong input " << endl;
    return 0;
  }


  srand (time(NULL));

  // tested on coPapersDBLP.mtx
  string filename = argv[1]; //"coPapersDBLP.mtx"; // edit
  //string filepath = "/home/cema/Documents/CS406/project/data/" + filename;
  ifstream file(filename.c_str());

  if(file.fail()) {
    cout << "Path not found" << endl;
    return 0;
  }

  string line;
  getline(file,line);

  stringstream s(line);
  bool symmetric = false;
  string word;
  while(s >> word) // check for symmetric
    if(word == "symmetric")
      symmetric = true;

  while(file.peek() == '%')//dirty boi
    file.ignore(2048, '\n');

  int  M, edge;
  file >> N >> M >> edge;



  cout << "Graph has " << N << " nodes and " << (1+symmetric*1)*edge << " edges and symmetric " << symmetric << endl;

  int n;
  bool based0 = false;
  while(file >> n) {
    if (n == 0) {
      based0 = true;
      break;
    }
  }
  file.close();




  if(based0)
    cout << "Graph is 0 based" << endl;
  else
    cout << "Graph is 1 based and is being turned in to 0 base." << endl;

  file.open(filename.c_str());
  while(file.peek() == '%')
    file.ignore(2048, '\n');
  file >> n >> n >> n; //tmp


  int i,j;

  vector<vector<int> > v(N, vector<int>(0));

  for(int k = 0; k < edge; k++) {
    file >> i >> j;
    if(!based0) { // make 0 based;
      i--;
      j--;
    }

    v[i].push_back(j);
    if(symmetric)
      v[j].push_back(i);

  }
  file.close();

  edge *= 1 + (int)symmetric;


  row = new int[N+1];
  col = new int[edge];

  row[0] = 0;
  int index = 0;
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < v[i].size(); j++) {
      col[index] = v[i][j];
      index++;
    }
    row[i+1] = index;
  }

  path = new int[N];
  visited = new bool[N];
  //    int *levels = new int[N];
  for(int i = 0; i < N; i++)
    path[i] =  INT_MAX;
 

 
 cout << "Preprocessing complete " << endl;


  // algorithm starts here
  //*************************************************
  hipSetDevice(0);
  int root = rand() % (N-1);
  root = 0;
  path[root] = 0;

  int root_begin = row[root];
  int root_end = row[root+1];
  
  for(int I = root_begin; I < root_end; I++){
	path[col[I]] = 1;
}
  // copy graph to device
  int *d_row;
  int *d_col;
  int *d_path;
  int *d_load;
  int size_row = (N+1)*sizeof(int);
  int size_col = edge*sizeof(int);
  int size_path = N*sizeof(int);
  int load = 1;

  // for the row check up
  hipMalloc((void **)&d_row,size_row);
  hipMalloc((void **)&d_col,size_col);
  // path array
  hipMalloc((void **)&d_path,size_path);
  // integer for check operation
  hipMalloc((void **)d_load,sizeof(int)); 
  
  cout << "Memory allocated" <<endl;
  // move memory
  hipMemcpy(d_row,row,size_row,hipMemcpyHostToDevice);
  hipMemcpy(d_col,col,size_col,hipMemcpyHostToDevice);
  hipMemcpy(d_path,path,size_path,hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  cout << "Graph copied to device" << endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  unsigned int threads = prop.maxThreadsPerBlock;
  unsigned int threadDIM = prop.maxThreadsDim[3];

  cout << "Here are the specs\n";
  cout << "The threads: " << threads << "\n";
  cout << "Thread DIM:  " << threadDIM << "\n";
  int BlockDIM = int(ceil(double(N)/threads));  // BlockDIM ->> number of blocks necessary

  int level = 1;
  cout << "Algo starts now... Hold on to your seats" << endl;
  auto start = omp_get_wtime();

  while(load != 0) {
    top_down_step<<<BlockDIM,threads>>>(d_load,path,level,d_row,d_col);
    hipDeviceSynchronize();
    hipMemcpy(&load,d_load,sizeof(int),hipMemcpyDeviceToHost);
    cout << "LOAD: " << load << endl;
    hipDeviceSynchronize();
    level++;
  }
  auto finish = omp_get_wtime();


  hipMemcpy(path,d_path,size_path,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  int count = 0;
  for(int i = 0; i < N; i++) {
    if(path[i] == INT_MAX) {
      count++;
      //cout << i << endl;
    }
  }

  if(count == 0)
    cout << "BFS Complete" << endl;
  else
    cout << count << " unvisited nodes" << endl;

  auto elapsed = finish - start;
  cout << "Time: " << elapsed << endl;

  hipFree(d_row);
  hipFree(d_col);
  hipFree(d_path);

  delete[] col;
  delete[] row;
  //delete[] path;

  cout << "Graph deleted" << endl;

  ofstream ofile;
  ofile.open("results.txt");
  for(int i = 0; i < N; i++)
    ofile << path[i] << "\n";
  ofile.close();

  delete[] path;
  return 0;
}
